#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __host__ __device__ void collect_bucket<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, oracle bucket, index* atomic);
template __host__ __device__ void collect_bucket_indirect<float,select_config<10, 10, 8, true, false, true, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, const oracle* bucket, index* atomic);
template __host__ __device__ launch_parameters get_launch_parameters<float,select_config<10, 10, 8, false, false, false, 8, 10, 10>>(index size);
template void sampleselect<double,select_config<10, 10, 6, true, false, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect<float,select_config<10, 10, 7, true, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template __global__ void kernels::count_buckets<float, select_config<10, 12, 10, true, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template __device__ __host__ void partition<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* in, float* out, index* counts, index size, float pivot);
template __global__ void kernels::partition<float,select_config<10, 5, 8, false, true, true, 8, 10, 10>>(const float* in, float* out, index* atomic, index size, float pivot, index workcount);
template __host__ __device__ void collect_bucket<float,select_config<10, 12, 10, false, true, false, 8, 10, 10>>(const float* data, const poracle* oracles_packed, const index* prefix_sum, float* out, index size, oracle bucket, index* atomic);
template __host__ __device__ void collect_bucket_indirect<double,select_config<10, 10, 8, true, false, true, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, const oracle* bucket, index* atomic);
}