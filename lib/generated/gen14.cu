#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __host__ __device__ void build_searchtree<float,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(const float* in, float* out, index size);
template void sampleselect_multi<double,select_config<10, 10, 8, false, true, true, 2, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template __global__ void kernels::reduce_counts<select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const index* in, index* out, index);
template void sampleselect<double,select_config<10, 10, 6, false, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect<double,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 4>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __global__ void kernels::count_buckets<double, select_config<10, 11, 9, false, true, false, 8, 10, 10>>(const double* in, const double* tree, index* counts, poracle* oracles, index size, index workcount);
template __device__ void kernels::masked_prefix_sum<9>(index* counts, const mask* m);
template __host__ __device__ void collect_bucket<double,select_config<10, 11, 9, false, true, false, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template __global__ void kernels::prefix_sum_counts<select_config<10, 10, 8, true, true, true, 8, 10, 10>>(index* in, index* out, index);
template void sampleselect_multi<double,select_config<10, 10, 6, false, false, true, 8, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
}