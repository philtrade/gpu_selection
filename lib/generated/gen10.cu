#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template void quickselect<float,select_config<9, 10, 8, false, true, true, 8, 10, 10>>(float* in, float* tmp, index* count_tmp, index size, index rank, float* out);
template void sampleselect<double,select_config<9, 10, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ launch_parameters get_launch_parameters<double,select_config<10, 8, 6, false, true, false, 8, 10, 10>>(index size);
template __global__ void kernels::reduce_counts<select_config<10, 9, 7, true, true, false, 8, 10, 10>>(const index* in, index* out, index);
template __host__ __device__ void collect_bucket<double,select_config<10, 10, 8, true, false, false, 8, 10, 10>>(const double* data, const poracle* oracles_packed, const index* prefix_sum, double* out, index size, oracle bucket, index* atomic);
template void quickselect_multi<double,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, const index* ranks, index rank_count, double* out);
template void sampleselect_multi<float,select_config<10, 10, 8, true, true, true, 8, 10, 10, false, 8, 0>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template __host__ __device__ void build_searchtree<float,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(const float* in, float* out, index size);
template void sampleselect_multi<float,select_config<10, 10, 6, false, true, true, 8, 10, 10>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void quickselect<double,select_config<10, 10, 8, false, true, true, 8, 8, 10>>(double* in, double* tmp, index* count_tmp, index size, index rank, double* out);
}