#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template void sampleselect<float,select_config<10, 10, 6, false, true, true, 8, 10, 10>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template void sampleselect_multi<double,select_config<10, 10, 8, true, true, true, 8, 10, 10, false, 8, 0>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template void sampleselect<double,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 16>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ launch_parameters get_launch_parameters<float,select_config<10, 12, 10, true, true, false, 8, 10, 10>>(index size);
template void sampleselect<double,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template void sampleselect_multi<float,select_config<10, 10, 8, false, true, true, 8, 10, 10, false, 16>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template __global__ void partition_count<double,select_config<10, 5, 8, true, true, true, 8, 10, 10>>(const double* in, index* counts, index size, double pivot, index workcount);
template void quickselect<double,select_config<10, 10, 8, false, true, true, 8, 10, 10>>(double* in, double* tmp, index* count_tmp, index size, index rank, double* out);
template __global__ void kernels::count_buckets<float, select_config<10, 9, 7, false, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
}