#include "hip/hip_runtime.h"
#include <kernel_config.cuh>
#include <qs_launchers.cuh>
#include <qs_recursion.cuh>
#include <qs_recursion_multi.cuh>
#include <ssss_recursion.cuh>
#include <ssss_recursion_multi.cuh>
#include <ssss_launchers.cuh>
namespace gpu {
template __global__ void kernels::count_buckets<float, select_config<10, 12, 10, false, true, false, 8, 10, 10>>(const float* in, const float* tree, index* counts, poracle* oracles, index size, index workcount);
template void sampleselect<float,select_config<10, 10, 6, false, true, true, 8, 10, 10, false, 1024>>(float* in, float* tmp, float* tree, index* count_tmp, index size, index rank, float* out);
template __global__ void kernels::partition_distr<double,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const double* in, double* out, const index* counts, index size, double pivot, index workcount);
template void sampleselect_multi<double,select_config<10, 10, 6, false, true, true, 8, 10, 10, false, 1024>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
template void sampleselect<double,select_config<10, 10, 8, false, false, true, 8, 10, 10>>(double* in, double* tmp, double* tree, index* count_tmp, index size, index rank, double* out);
template __host__ __device__ void ssss_merged<float,select_config<10, 10, 8, true, true, true, 8, 10, 10>>(const float* in, float* out, poracle* oracles, index offset, const index* ranks, index rank_offset, index rank_base, const kernels::ssss_multi_aux<float, select_config<10, 10, 8, true, true, true, 8, 10, 10>>* aux_in, kernels::ssss_multi_aux<float, select_config<10, 10, 8, true, true, true, 8, 10, 10>>* aux_outs, float* out_tree);
template __global__ void kernels::prefix_sum_counts<select_config<10, 10, 8, false, true, false, 8, 10, 10>>(index* in, index* out, index);
template void sampleselect_multi<float,select_config<10, 10, 8, true, true, true, 8, 10, 10, false, 4>>(float* in, float* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, float* out);
template void sampleselect_multi<double,select_config<10, 10, 8, true, true, true, 2, 10, 10>>(double* in, double* tmp, index size, const index* ranks, index rank_count, index* tmp_storage, index* aux_storage, index* aux_atomic, double* out);
}